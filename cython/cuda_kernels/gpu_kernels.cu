#include "hip/hip_runtime.h"
#import <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#import "gpu_kernels.h"
#import "../utils/gpu_utils.cuh"

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__
void kernel() {
    printf("hello from the kernel!\n");
}

void gpuf() {
    printf("hello from the gpu file!\n");
    hipDeviceSynchronize();
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}


#define BLOCK_SIZE 256//1024

/// https://github.com/rapidsai/cuml/blob/branch-22.04/cpp/src/umap/runner.cuh

__device__
float sqrd_dist(const float *__restrict__ d_D, const int dims, const int i, const int j) {
    float distance = 0.;
    for (int l = 0; l < dims; l++) {
        float diff = d_D[i * dims + l] - d_D[j * dims + l];
        distance += diff * diff;
    }
    return distance;
}

__device__
float q(float distance) {
    return 1 / (1 + distance * distance);
}

__global__
void init_random(hiprandState *d_random) {
    //initialize d_random
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int seed = id; // different seed per thread
    hiprand_init(seed, id, 0, &d_random[id]);
}


__device__
int get_start(const int *d_ends, int i) {
    return i == 0 ? 0 : d_ends[i - 1];
}

__device__
int get_end(const int *d_ends, int i) {
    return d_ends[i];
}

__device__
double fast_pow(double a, double b) {
    union {
        double d;
        int x[2];
    } u = {a};
    if (b == 1.0) {
        return a;
    }
    u.x[1] = (int) (b * (u.x[1] - 1072632447) + 1072632447);
    u.x[0] = 0;
    return u.d;
}

__device__
float kernel_function(float dist_squared, float a, float b) {
    if (b <= 1)
        return 1 / (1 + a * fast_pow(dist_squared, b));
    return fast_pow(dist_squared, b - 1) / (1 + a * fast_pow(dist_squared, b));
}

__device__
float umap_attr_scalar(float dist_squared, float a, float b) {
    float grad_scalar = 0.0;
    grad_scalar = 2.0 * a * b * fast_pow(dist_squared, b - 1.0);
    grad_scalar /= a * fast_pow(dist_squared, b) + 1.0;
    return grad_scalar;
}

__device__
float frob_attr_force(int normalized, float p, float q) {
    if (normalized) {
        // FIXME - is it faster to get q^2 and then use that for q^3?
        // FIXME - took out a Z scalar from this
        return p * (q * q + 2 * pow(q, 3));
    }
    return p * q * q;
}

__device__
float kl_attr_force(float p, float q) {
    return p * q;
}

__device__
float attractive_force_func(
        int frob,
        int normalized,
        float dist_squared,
        float a,
        float b,
        float edge_weight
) {

    float q;
    if (normalized || frob)
        q = kernel_function(dist_squared, a, b);
    else
        q = umap_attr_scalar(dist_squared, a, b);

    if (frob)
        return frob_attr_force(normalized, edge_weight, q);
    else
        return kl_attr_force(edge_weight, q);


}
__device__
float norm_rep_force(
        float *d_Z,
        int i_thread,
        float dist_squared,
        float a,
        float b,
        float cell_size
) {
    float kernel, q_ij, repulsive_force;

    kernel = kernel_function(dist_squared, a, b);

    q_ij = cell_size * kernel; // Collect the q_ij's contributions into Z
    d_Z[i_thread] += q_ij;

    repulsive_force = cell_size * kernel * kernel;
    return repulsive_force;
}

__device__
float umap_rep_scalar(float dist_squared, float a, float b) {
    float phi_ijZ = 0.0;
    phi_ijZ = 2.0 * b;
    phi_ijZ /= (0.001 + dist_squared) * (a * fast_pow(dist_squared, b) + 1);
    return phi_ijZ;
}

__device__
float frob_rep_force(int normalized, float q) {
    if (normalized)
        return pow(q, 3) + 2 * pow(q, 4);
    return pow(q, 3);
}

__device__
float kl_rep_force(int normalized, float q, float avg_weight) {
    if (normalized)
        return q * q;
    return q * (1 - avg_weight);
}

__device__
float repulsive_force_func(
        float *d_Z,
        int i_thread,
        int frob,
        int normalized,
        float dist_squared,
        float a,
        float b,
        float cell_size,
        float average_weight
) {
    float q, result;
    if (normalized || frob)
        q = kernel_function(dist_squared, a, b);
    else
        q = umap_rep_scalar(dist_squared, a, b);

    if (frob)
        result = frob_rep_force(normalized, q);
    else
        result = kl_rep_force(normalized, q, average_weight);
    result *= cell_size;

    if (normalized)
        d_Z[i_thread] += q * cell_size;

    return result;
}

__global__
void
compute_grads_full_shared_mem_N(const int frob, const int normalized, float *__restrict__ d_rep_grads,
                                float *__restrict__ d_attr_grads,
                                const float *__restrict__ d_weights,
                                const int n,
                                const int *__restrict__ d_N,
                                const int *__restrict__ d_neighbor_ends,
                                const float *__restrict__ d_D_embed,
                                float *__restrict__ d_Z, const float a, const float b, const int dims_embed,
                                hiprandState *__restrict__ d_random, const float sym_attraction,
                                const float weight_scalar,
                                const float average_weight, const int k, const int negative_sample_rate) {

    extern __shared__ float s_rep_grads[];
    float *s_attr_grads = &s_rep_grads[blockDim.x * dims_embed];

    int i_thread = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    for (int i_point = threadIdx.x + blockIdx.x * blockDim.x; i_point < n; i_point += blockDim.x * gridDim.x) {

        for (int h = 0; h < dims_embed; h++) {
            s_rep_grads[tid * dims_embed + h] = 0.;
            s_attr_grads[tid * dims_embed + h] = 0.;
        }

        for (int i_edge = get_start(d_neighbor_ends, i_point); i_edge < get_end(d_neighbor_ends, i_point); i_edge++) {
            int j_point = d_N[i_edge];

            float dist_squared = sqrd_dist(d_D_embed, dims_embed, i_point, j_point);
            float attr = attractive_force_func(
                    frob,
                    normalized,
                    dist_squared,
                    a,
                    b,
                    d_weights[i_edge] * weight_scalar
            );
            for (int h = 0; h < dims_embed; h++) {

                float force = attr * (d_D_embed[i_point * dims_embed + h] - d_D_embed[j_point * dims_embed + h]);
                s_attr_grads[tid * dims_embed + h] -= force;
                atomicAdd(&d_attr_grads[j_point * dims_embed + h],
                          force * sym_attraction);
            }

            int g = hiprand(&d_random[i_thread]) % n;//random int
            dist_squared = sqrd_dist(d_D_embed, dims_embed, i_point, g);
            float rep = repulsive_force_func(
                    d_Z,
                    i_thread,
                    frob,
                    normalized,
                    dist_squared,
                    a,
                    b,
                    1.0,
                    average_weight
            );

            for (int h = 0; h < dims_embed; h++) {
                s_rep_grads[tid * dims_embed + h] +=
                        rep * (d_D_embed[i_point * dims_embed + h] - d_D_embed[g * dims_embed + h]);
            }


        }

        for (int h = 0; h < dims_embed; h++) {

            atomicAdd(&d_rep_grads[i_point * dims_embed + h], s_rep_grads[tid * dims_embed + h]);
            atomicAdd(&d_attr_grads[i_point * dims_embed + h], s_attr_grads[tid * dims_embed + h]);

        }
    }
}

__device__
float clip(float val, float lower, float upper) {
    return fmax(lower, fmin(val, upper));
}

__global__
void
apply_grads_full(float *d_Z, float *d_D_embed, float *d_rep_grads, float *d_attr_grads, float *d_all_grads,
                 float *d_gains,
                 int n, int dims_embed, float lr, float a, float b, float amplify_graps) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x) {
        for (int h = 0; h < dims_embed; h++) {
            int index = i * dims_embed + h;
            float grad = (d_rep_grads[index] / d_Z[0] + d_attr_grads[index]) * 4 * a * b;

            if (grad * d_all_grads[index] > 0.0)
                d_gains[index] += 0.2;
            else
                d_gains[index] *= 0.8;
            d_gains[index] = clip(d_gains[index], 0.01, 100);
            grad *= d_gains[index];

            d_all_grads[index] *= (float) amplify_graps * 0.9;
            d_all_grads[index] += grad * lr;

            d_D_embed[index] += d_all_grads[index];
        }
    }
}

float get_lr(float initial_lr, int i_epoch, int n_epochs, int amplify_graps) {
    if (amplify_graps)
        return initial_lr;
    return initial_lr * (1.0 - (((float) i_epoch) / ((float) n_epochs)));
}

__global__
void convert(int *d_dst_int, long *d_src_long, int n) {

    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x) {
        d_dst_int[i] = (int) d_src_long[i];
    }
}

__global__
void reduced_sum_fix(float *d_out, float *d_in, int n) {
    extern __shared__ float s_tmp[];
    int i_thread = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    s_tmp[tid] = 0;
    for (int s = 0; s < n; s += gridDim.x * blockDim.x) {
        if (i_thread + s < n) {
            s_tmp[tid] += d_in[i_thread + s];
        }
    }
    d_out[i_thread] = s_tmp[tid];
}

__global__
void reduced_sum(float *d_out, float *d_in, int n) {

    extern __shared__ float s_tmp[];

    int i_thread = threadIdx.x + blockIdx.x * (blockDim.x * 2);
    int tid = threadIdx.x;

    int n_active_threads = blockDim.x;

    s_tmp[tid] = 0;
    if (i_thread < n)
        s_tmp[tid] += d_in[i_thread];
    if (i_thread + blockDim.x < n)
        s_tmp[tid] += d_in[i_thread + blockDim.x];
    __syncthreads();

    for (int n_active_threads = blockDim.x / 2; n_active_threads > 0; n_active_threads >>= 1) {
        if (tid < n_active_threads) {
            s_tmp[tid] += s_tmp[tid + n_active_threads];
            __syncthreads();
        }
    }

    if (tid == 0) d_out[blockIdx.x] = s_tmp[0];
}

float mean(float *h_x, int n) {
    float x = 0;
    for (int i = 0; i < n; i++) {
        x += h_x[i];
    }
    return x / n;
}

__global__
void compute_max(int *d_out, int *d_in, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x) {
        atomicMax(&d_out[0], d_in[i]);
    }
}

int gpu_max(int *d_in, int n) {
    int *d_out = gpu_malloc_int_zero(1);
    compute_max<<<32, BLOCK_SIZE>>>(d_out, d_in, n);
    int m = copy_last_D_to_H(d_out, 1);
    hipFree(d_out);
    return m;
}

__global__
void pack_N(int *d_N_new, float *d_weights_new, int *d_N, float *d_weights, int *d_neighbor_ends, int n_vertices,
            int k) {
    for (int i_point = threadIdx.x + blockIdx.x * blockDim.x; i_point < n_vertices; i_point += blockDim.x * gridDim.x) {
        int loc = 0;
        for (int i_edge = get_start(d_neighbor_ends, i_point); i_edge < get_end(d_neighbor_ends, i_point); i_edge++) {
            d_N_new[i_point * k + loc] = d_N[i_edge];
            d_weights_new[i_point * k + loc] = d_weights[i_edge];
            loc++;
        }
    }
}


void gpu_umap_full_N(int normalized, // unused
                     int sym_attraction, // unused
                     int frob,
                     int amplify_graps, // unused
                     float *h_D_embed, //head_embedding,
                     float *h_D_embed_other, //tail_embedding,
                     int *h_N, //head,
                     int *tail, // im not using this
                     float *h_weights,//weights,
                     long *h_neighbor_counts, //neighbor_counts,
                     float *all_updates, // unused
                     float *gains, // unused
                     float a, // unused
                     float b, // unused
                     int dims_embed, //dim,
                     int n_vertices,
                     float init_lr,
                     int n_epochs,
                     int n_edges,
                     int negative_sample_rate
) {
    hipDeviceSynchronize();
    int number_of_blocks_scalar = 32;//32 can be replace with something smaller then BLOCK_SIZE
    int number_of_threads_in_total = BLOCK_SIZE * 2 * number_of_blocks_scalar;

    //allocated and copy memory to the gpu
    float *d_D_embed = copy_H_to_D(h_D_embed, n_vertices * dims_embed);
    int *d_N = copy_H_to_D(h_N, n_edges);
    long *d_neighbor_counts_long = copy_H_to_D(h_neighbor_counts, n_vertices);
    int *d_neighbor_counts = gpu_malloc_int(n_vertices);
    int *d_neighbor_ends = gpu_malloc_int_zero(n_vertices);
    float *d_weights = copy_H_to_D(h_weights, n_edges);
    float *d_rep_grads = gpu_malloc_float(n_vertices * dims_embed);
    float *d_attr_grads = gpu_malloc_float(n_vertices * dims_embed);
    float *d_all_grads = gpu_malloc_float_zero(n_vertices * dims_embed);
    float *d_gains = gpu_malloc_float(n_vertices * dims_embed);
    gpu_set_all(d_gains, n_vertices * dims_embed, 1.);
    float *d_Z = gpu_malloc_float(number_of_threads_in_total);

    float *d_tmp_sum_1 = gpu_malloc_float(number_of_threads_in_total);
    float *d_tmp_sum_2 = gpu_malloc_float(number_of_blocks_scalar);


    int number_of_threads = min(n_vertices, number_of_threads_in_total);
    int number_of_blocks = number_of_threads / BLOCK_SIZE;
    if (number_of_threads % BLOCK_SIZE) number_of_blocks++;

    int number_of_blocks_half = (number_of_threads_in_total / 2) / BLOCK_SIZE;
    if ((number_of_threads_in_total / 2) % BLOCK_SIZE) number_of_blocks_half++;


    //random
    hiprandState *d_random;
    hipMalloc((void **) &d_random, number_of_threads * sizeof(hiprandState));
    init_random << < number_of_blocks, BLOCK_SIZE >> > (d_random);

    convert<<<number_of_blocks, BLOCK_SIZE>>>(d_neighbor_counts, d_neighbor_counts_long, n_vertices);
    inclusive_scan(d_neighbor_counts, d_neighbor_ends, n_vertices);

    int k = gpu_max(d_neighbor_counts, n_vertices);

    reduced_sum_fix<<<number_of_blocks_half * 2, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>
            (d_tmp_sum_1, d_weights, n_edges);
    reduced_sum<<<number_of_blocks_half, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>
            (d_tmp_sum_2, d_tmp_sum_1, number_of_threads_in_total);
    reduced_sum<<<1, number_of_blocks_scalar, number_of_blocks_scalar * sizeof(float)>>>
            (d_tmp_sum_1, d_tmp_sum_2, number_of_blocks_scalar);
    float average_weight = copy_last_D_to_H(d_tmp_sum_1, 1) / n_edges;

//    printf("\n\nParams:\n");
//    printf("- average_weight: %f\n", average_weight);
//    printf("- amplify_graps: %d\n", amplify_graps);
//    printf("- sym_attraction: %d\n", sym_attraction);
//    printf("- normalized: %d\n", normalized);
//    printf("- n_edges: %d\n", n_edges);
//    printf("- negative_sample_rate: %d\n", negative_sample_rate);
//    printf("- a: %f\n", a);
//    printf("- b: %f\n", b);
//    printf("- number_of_blocks_scalar: %d\n", number_of_blocks_scalar);
//    printf("- number_of_blocks_half: %d\n", number_of_blocks_half);
//    printf("- number_of_blocks: %d\n", number_of_blocks);
//    printf("\n\n");

    gpu_set_all(d_tmp_sum_2, 1, 1.);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    for (int i_epoch = 0; i_epoch < n_epochs; i_epoch++) {
        float lr = get_lr(init_lr, i_epoch, n_epochs, amplify_graps);
        hipMemset(d_rep_grads, 0, n_vertices * dims_embed * sizeof(float));
        hipMemset(d_attr_grads, 0, n_vertices * dims_embed * sizeof(float));
        hipMemset(d_Z, 0, number_of_threads_in_total * sizeof(float));
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        float weight_scalar;
        if (i_epoch < 100)
            weight_scalar = 4;
        else
            weight_scalar = 1;

        int number_of_blocks_n = n_vertices/BLOCK_SIZE;
        if (n_vertices % BLOCK_SIZE) number_of_blocks++;

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        compute_grads_full_shared_mem_N <<<
        number_of_blocks,
        BLOCK_SIZE,
        BLOCK_SIZE * dims_embed * 2 * sizeof(float)>>>
                (frob, normalized, d_rep_grads, d_attr_grads,
                 d_weights,
                 n_vertices,
                 d_N, d_neighbor_ends,
                 d_D_embed, d_Z,
                 a, b, dims_embed, d_random, sym_attraction, weight_scalar, average_weight, k, negative_sample_rate);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        float Z = 0.;
        if (normalized) {
            reduced_sum<<<number_of_blocks_half, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>
                    (d_tmp_sum_1, d_Z, number_of_threads_in_total);
            reduced_sum<<<1, number_of_blocks_scalar, number_of_blocks_scalar * sizeof(float)>>>
                    (d_tmp_sum_2, d_tmp_sum_1, number_of_blocks_scalar);
        }

        apply_grads_full << < number_of_blocks, BLOCK_SIZE>> >
        (d_tmp_sum_2, d_D_embed, d_rep_grads, d_attr_grads, d_all_grads, d_gains, n_vertices, dims_embed, lr, a, b, amplify_graps);


        if ((i_epoch + 1) % 50 == 0) {
            printf("Epoch %d/%d\n", i_epoch + 1, n_epochs);
        }

    }

    //copy back and delete
    hipMemcpy(h_D_embed, d_D_embed, n_vertices * dims_embed * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_D_embed);
    hipFree(d_N);
    hipFree(d_neighbor_counts_long);
    hipFree(d_neighbor_counts);
    hipFree(d_neighbor_ends);
    hipFree(d_weights);
    hipFree(d_rep_grads);
    hipFree(d_attr_grads);
    hipFree(d_all_grads);
    hipFree(d_gains);
    hipFree(d_random);
    hipFree(d_Z);
    hipFree(d_tmp_sum_1);
    hipFree(d_tmp_sum_2);
}


void gpu_umap(
        int normalized,
        int sym_attraction,
        int frob,
        int amplify_graps,
        float *head_embedding,
        float *tail_embedding,
        int *head,
        int *tail,
        float *weights,
        long *neighbor_counts,
        float *all_updates,
        float *gains,
        float a,
        float b,
        int dim,
        int n_vertices,
        float initial_lr,
        int n_edges,
        int n_epochs,
        int negative_sample_rate
) {
    int k = n_edges / n_vertices;
    gpu_umap_full_N(
            normalized, // unused
            sym_attraction, // unused
            frob,
            amplify_graps, // unused
            head_embedding,
            tail_embedding,
            head,
            tail,
            weights,
            neighbor_counts,
            all_updates, // unused
            gains, // unused
            a, // unused
            b, // unused
            dim,
            n_vertices,
            initial_lr,
            n_epochs,
            n_edges,
            negative_sample_rate
    );
}



__global__
void KNN(int *d_neighbors, float *d_distances, float *d_data, int n, int d, int k) {
    extern __shared__ float s_array[];
    float *s_distances = &s_array[2 * k * threadIdx.x];
    int *s_neighbors = (int *) &s_distances[k];
    for (int i_point = threadIdx.x + blockIdx.x * blockDim.x; i_point < n; i_point += blockDim.x * gridDim.x) {

        for (int i = 0; i < k; i++) {
            s_distances[i] = 0.;
        }

        for (int j_point = 0; j_point < n; j_point) {
            float distance = 0.;
            for (int i_dim = 0; i_dim < d; i_dim++) {
                float diff = d_data[i_point * d + i_dim] - d_data[j_point * d + i_dim];
                distance += diff * diff;
            }
            distance = sqrt(distance);
        }
    }
}

void GPU_KNN(int *h_neighbors, float *h_distances, float *h_data, int n, int d, int k) {


    ///test
    int *d_neighbors = gpu_malloc_int(n * k);
    float *d_distances = gpu_malloc_float(n * k);
    float *d_data = copy_H_to_D(h_data, n * d);
    int number_of_blocks = n / BLOCK_SIZE;
    if (n % BLOCK_SIZE)number_of_blocks++;
    KNN<<<number_of_blocks, BLOCK_SIZE>>>(d_neighbors, d_distances, d_data, n, d, k);
}