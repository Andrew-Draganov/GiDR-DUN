#include "hip/hip_runtime.h"
#import <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#import "gpu_kernels.h"


__global__
void kernel(){
    printf("hello from the kernel!\n");
}

void gpuf(){
    printf("hello from the gpu file!\n");
    hipDeviceSynchronize();
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
}